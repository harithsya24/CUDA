#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define N 1000000
#define BLOCK_SIZE 256

void vector_add_cpu(float *a, float *b, float *c, int n) {
	for (int i = 0; i < n; i++) {
		c[i] = a[i] + b[i];
	}
}
__global__ void vector_add_gpu(float *a, float *b, float *c, int n) {
	int i =blockIdx.x * blockDim.x  + threadIdx.x;
	if (i <n) {
		c[i] = a[i] + b[i];
	}
}

void init_vector(float *vec, int n){
	for (int i = 0; i<n; i++){
		vec[i] = (float)rand() / RAND_MAX;
	}
}

double get_time(){
	struct timespec ts;
	clock_gettime(CLOCK_MONOTONIC, &ts);
	return ts.tv_sec + ts.tv_nsec * 1e-9;
}

int main(){
	float *h_a, *h_b, *h_c_cpu, *h_c_gpu;
	float *d_a, *d_b, *d_c;
	size_t size = N *sizeof(float);

	h_a = (float*)malloc(size);
	h_b = (float*)malloc(size);
	h_c_cpu = (float*)malloc(size);
	h_c_gpu = (float*)malloc(size);

	srand(time(NULL));
	init_vector(h_a, N);
	init_vector(h_b, N);

	hipMalloc(&d_a, size);
	hipMalloc(&d_b, size);
	hipMalloc(&d_c, size);

	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

	int num_blocks = (N +BLOCK_SIZE -1) /BLOCK_SIZE;

	printf("Performing warm-up runs...\n");
	for (int i = 0; i < 3; i++) {
		vector_add_cpu(h_a, h_b, h_c_cpu, N);
		vector_add_gpu<<<num_blocks,BLOCK_SIZE>>>(d_a, d_b, d_c, N);
		hipDeviceSynchronize();
	}

	printf("Benchmarking CPU implementation..\n");
	double cpu_total_time = 0.0;
	for(int i= 0; i < 20; i++) {
		double start_time = get_time();
		vector_add_cpu(h_a, h_b, h_c_cpu, N);
		double end_time = get_time();
		cpu_total_time += end_time - start_time;
	}
	double cpu_avg_time = cpu_total_time /20.0;

	printf("Benchmarking GPU implementation..\n");
	double gpu_total_time = 0.0;
	for (int i=0; i < 20; i++){
		double start_time = get_time();
		vector_add_gpu<<<num_blocks, BLOCK_SIZE>>>(d_a, d_b, d_c, N);
		hipDeviceSynchronize();
		double end_time = get_time();
		gpu_total_time += end_time -start_time;
	}
	double gpu_avg_time = gpu_total_time /20.0;

	printf("CPU avergae time  = %f milliseconds\n", cpu_avg_time*1000);
	printf("GPU avergae time = %f milliseconds\n", gpu_avg_time*1000);
	printf("Speedup: %fx\n", cpu_avg_time /gpu_avg_time);

	hipMemcpy(h_c_gpu, d_c, size, hipMemcpyDeviceToHost);
	bool correct = true;
	for( int i= 0; i<N; i++){
		if(fabs(h_c_cpu[i] - h_c_gpu[i]) > 1e-5){
			correct = false;
			break;
		}
	}
	printf("Results are %s\n", correct ? "correct" : "incorrect");

	free(h_a);
	free(h_b);
	free(h_c_cpu);
	free(h_c_gpu);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
